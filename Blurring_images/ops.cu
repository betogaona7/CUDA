#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void gaussian_blur(const unsigned char* const inputChannel,
                              unsigned char* const outputChannel,
                              int numRows, int numCols,
                              const float* const filter, const int filterWidth){
  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                       blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  if(thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  float result = 0.0f;
  for(int r = -filterWidth/2; r <= filterWidth/2; ++r){
    for (int c = -filterWidth/2; c <= filterWidth/2; ++c){
        int row = min(max(0, thread_2D_pos.y + r), numRows - 1);
        int col = min(max(0, thread_2D_pos.x + c), numCols - 1);
        int pixel = row * numCols + col;
        result += float(filter[(r + filterWidth/2) * filterWidth + (col + filterWidth/2)] * inputChannel[pixel]);
    }
  }
  outputChannel[thread_1D_pos] = result;
}

__global__ void separateChannels(const uchar4* const inputImageRGBA,
                                unsigned char* const redChannel,
                                unsigned char* const greenChannel,
                                unsigned char* const blueChannel,
                                int numRows, int numCols){
  /* 
    Separates the different color channels so that each color is stored contiguously 
    instead of being interleaved. From an AoS to a SoA. 
  */
  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                       blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  if(thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
  greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
  blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;
}

__global__ void recombineChannels(const unsigned char* const redChannel,
                                  const unsigned char* const greenChannel,
                                  const unsigned char* const blueChannel,
                                  uchar4* const outputImageRGBA,
                                  int numRows, int numCols){
  /*
    Combines the different color channels. From a SoA to an AoS. 
  */
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  // alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);
  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  // allocate memory for the three different channels
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  // allocate memory for the filter on the GPU
  checkCudaErrors(hipMalloc(&d_filter, sizeof(float) * filterWidth * filterWidth));

  // copy the filter on the host to the GPU
  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
}



void apply_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                         uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                         unsigned char *d_redBlurred, 
                         unsigned char *d_greenBlurred, 
                         unsigned char *d_blueBlurred,
                         const int filterWidth){

  const dim3 blockSize(16, 16);
  const dim3 gridSize(numCols/blockSize.x +1, numRows/blockSize.y + 1);

  // Launch a kernel for separating the RGBA images
  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Call convolution kernel three times, once for each color channel.
  gaussian_blur<<<gridSize, blockSize>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  gaussian_blur<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());  

  gaussian_blur<<<gridSize, blockSize>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Recombine results
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred, d_greenBlurred, d_blueBlurred, d_outputImageRGBA, numRows, numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}
